#include "hip/hip_runtime.h"
/*
* TP 2 - Convolution d'images
* --------------------------
* Mémoire constante et textures
*
* File: student.cu
* Author: Maxime MARIA
*/
#include "student.hpp"
#include "chronoGPU.hpp"

namespace IMAC
{

		__constant__ float dev_matConv[1024];

		texture <uchar4, 1, hipReadModeElementType> dev_img_1D;
		texture <uchar4, 2, hipReadModeElementType> dev_img_2D;

// ================================================== For image comparison
	std::ostream &operator <<(std::ostream &os, const uchar4 &c)
	{
		os << "[" << uint(c.x) << "," << uint(c.y) << "," << uint(c.z) << "," << uint(c.w) << "]";
    	return os;
	}

	void compareImages(const std::vector<uchar4> &a, const std::vector<uchar4> &b)
	{
		bool error = false;
		if (a.size() != b.size())
		{
			std::cout << "Size is different !" << std::endl;
			error = true;
		}
		else
		{
			for (uint i = 0; i < a.size(); ++i)
			{
				// Floating precision can cause small difference between host and device
				if (	std::abs(a[i].x - b[i].x) > 2 || std::abs(a[i].y - b[i].y) > 2
					|| std::abs(a[i].z - b[i].z) > 2 || std::abs(a[i].w - b[i].w) > 2)
				{
					std::cout << "Error at index " << i << ": a = " << a[i] << " - b = " << b[i] << " - " << std::abs(a[i].x - b[i].x) << std::endl;
					error = true;
					break;
				}
			}
		}
		if (error)
		{
			std::cout << " -> You failed, retry!" << std::endl;
		}
		else
		{
			std::cout << " -> Well done!" << std::endl;
		}
	}

// ==================================================
__global__ void conv2D_CUDA(const uint imgWidth, const uint imgHeight, const uint matSize, uchar4* const dev_output)
{

		int idX = ((blockIdx.x * blockDim.x) + threadIdx.x);
		int idY = ((blockIdx.y * blockDim.y) + threadIdx.y);

		if(idX >= imgWidth || idY >= imgHeight)
		{
			return;
		}

		float3 sum = make_float3(0.f,0.f,0.f);
		int id = idY * imgWidth + idX;
		int i, j, x, y;
		uint idMat;
		uchar4 pixel;

		// Apply convolution
		for (j = 0; j < matSize; ++j )
		{
			for (i = 0; i < matSize; ++i )
			{
				x = min(imgWidth  - 1,  max(0, (idX + i - (int) matSize / 2)  ));
				y = min(imgHeight - 1,  max(0, (idY + j - (int) matSize / 2)  ));

				idMat		= j * matSize + i;
				//idImage	= y * imgWidth + x;

				pixel = tex2D(dev_img_2D, x, y);
				sum.x += (float) pixel.x * dev_matConv[idMat];
				sum.y += (float) pixel.y * dev_matConv[idMat];
				sum.z += (float) pixel.z * dev_matConv[idMat];
			}
		}

		dev_output[id].x = (uchar)min(255.f,  max(0.f, sum.x)  );
		dev_output[id].y = (uchar)min(255.f,  max(0.f, sum.y)  );
		dev_output[id].z = (uchar)min(255.f,  max(0.f, sum.z)  );
		dev_output[id].w = 255;
}


__global__ void conv1D_CUDA(const uint imgWidth, const uint imgHeight, const uint matSize, uchar4* const dev_output)
{
		int idX = ((blockIdx.x * blockDim.x) + threadIdx.x);
		int idY = ((blockIdx.y * blockDim.y) + threadIdx.y);

		if(idX >= imgWidth || idY >= imgHeight)
		{
			return;
		}

		float3 sum = make_float3(0.f,0.f,0.f);
		int id = idY * imgWidth + idX;
		int i, j, x, y;
		uint idMat, idImage;
		uchar4 pixel;

		// Apply convolution
		for (j = 0; j < matSize; ++j )
		{
			for (i = 0; i < matSize; ++i )
			{
				x = min(imgWidth  - 1,  max(0, (idX + i - (int) matSize / 2)  ));
				y = min(imgHeight - 1,  max(0, (idY + j - (int) matSize / 2)  ));

				idMat		= j * matSize + i;
				idImage	= y * imgWidth + x;

				pixel = tex1Dfetch(dev_img_1D, idImage);
				sum.x += (float) pixel.x * dev_matConv[idMat];
				sum.y += (float) pixel.y * dev_matConv[idMat];
				sum.z += (float) pixel.z * dev_matConv[idMat];
			}
		}

		dev_output[id].x = (uchar)min(255.f,  max(0.f, sum.x)  );
		dev_output[id].y = (uchar)min(255.f,  max(0.f, sum.y)  );
		dev_output[id].z = (uchar)min(255.f,  max(0.f, sum.z)  );
		dev_output[id].w = 255;
}

__global__ void convConstCUDA(	const uchar4* const dev_inputImg, const uint imgWidth, const uint imgHeight,
													      const uint matSize,
																uchar4* const dev_output)
{
		int idX = ((blockIdx.x * blockDim.x) + threadIdx.x);
		int idY = ((blockIdx.y * blockDim.y) + threadIdx.y);

		if(idX >= imgWidth || idY >= imgHeight)
		{
			return;
		}

		float3 sum = make_float3(0.f,0.f,0.f);
		int id = idY * imgWidth + idX;
		int i, j, x, y;
		uint idMat, idImage;
		uchar4 pixel;

		// Apply convolution
		for (j = 0; j < matSize; ++j )
		{
			for (i = 0; i < matSize; ++i )
			{
				x = min(imgWidth  - 1,  max(0, (idX + i - (int) matSize / 2)  ));
				y = min(imgHeight - 1,  max(0, (idY + j - (int) matSize / 2)  ));

				idMat		= j * matSize + i;
				idImage	= y * imgWidth + x;

				pixel = dev_inputImg[idImage];
				sum.x += (float) pixel.x * dev_matConv[idMat];
				sum.y += (float) pixel.y * dev_matConv[idMat];
				sum.z += (float) pixel.z * dev_matConv[idMat];
			}
		}

		dev_output[id].x = (uchar)min(255.f,  max(0.f, sum.x)  );
		dev_output[id].y = (uchar)min(255.f,  max(0.f, sum.y)  );
		dev_output[id].z = (uchar)min(255.f,  max(0.f, sum.z)  );
		dev_output[id].w = 255;
}

	__global__ void convCUDA(	const uchar4* const dev_inputImg, const uint imgWidth, const uint imgHeight,
														const float* const dev_matConv,  const uint matSize,
														uchar4* const dev_output)
{
			int idX = ((blockIdx.x * blockDim.x) + threadIdx.x);
			int idY = ((blockIdx.y * blockDim.y) + threadIdx.y);

			if(idX >= imgWidth || idY >= imgHeight)
			{
				return;
			}

			float3 sum = make_float3(0.f,0.f,0.f);
			int id = idY * imgWidth + idX;
			int i, j, x, y;
			uint idMat, idImage;
			uchar4 pixel;


			// Apply convolution
			for (j = 0; j < matSize; ++j )
			{
				for (i = 0; i < matSize; ++i )
				{
					x = min(imgWidth  - 1,  max(0, (idX + i - (int) matSize / 2)  ));
					y = min(imgHeight - 1,  max(0, (idY + j - (int) matSize / 2)  ));

					idMat		= j * matSize + i;
					idImage	= y * imgWidth + x;

					pixel = dev_inputImg[idImage];
					sum.x += (float)pixel.x * dev_matConv[idMat];
					sum.y += (float)pixel.y * dev_matConv[idMat];
					sum.z += (float)pixel.z * dev_matConv[idMat];
				}
			}

			dev_output[id].x = (uchar)min(255.f,  max(0.f, sum.x)  );
			dev_output[id].y = (uchar)min(255.f,  max(0.f, sum.y)  );
			dev_output[id].z = (uchar)min(255.f,  max(0.f, sum.z)  );
			dev_output[id].w = 255;
	}

  void studentJob(const std::vector<uchar4> &inputImg, // Input image
					const uint imgWidth, const uint imgHeight, // Image size
                    const std::vector<float> &matConv, // Convolution matrix (square)
					const uint matSize, // Matrix size (width or height)
					const std::vector<uchar4> &resultCPU, // Just for comparison
                    std::vector<uchar4> &output // Output image
					)
	{
		ChronoGPU chrGPU;

		uchar4 *dev_inputImg = NULL;
		uchar4 *dev_output   = NULL;

		const size_t imgBytes = imgWidth * imgHeight * sizeof(uchar4);
		const size_t matBytes = matSize  * matSize   * sizeof(uint);
		const int thread_size = 32;
		dim3 nb_block = dim3((imgWidth / thread_size) + 1, (imgHeight / thread_size) + 1,1);
		dim3 nb_thread = dim3(thread_size, thread_size, 1);

		////////////////////////////////    Exo 1   ////////////////////////////////////////
		/*
		float  *dev_matConv	 = NULL;

		chrGPU.start();
		hipMalloc((void **) &dev_inputImg, imgBytes);
		hipMalloc((void **) &dev_matConv,  matBytes);
		hipMalloc((void **) &dev_output,   imgBytes);
		chrGPU.stop();
		std::cout << "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;
		// Copy data from host to device (input arrays)
		hipMemcpy(dev_inputImg, &inputImg[0], imgBytes, hipMemcpyHostToDevice);
		hipMemcpy(dev_matConv,  &matConv[0],  matBytes, hipMemcpyHostToDevice);
		// Launch kernel
		convCUDA<<<nb_block, nb_thread>>>(dev_inputImg, imgWidth, imgHeight, dev_matConv, matSize, dev_output);
		// Copy data from device to host (output array)
		hipMemcpy(&output[0], dev_output, imgBytes, hipMemcpyDeviceToHost);
		// Free arrays on device
		hipFree(dev_matConv);
		*/
		////////////////////////////////////////////////////////////////////////////////////


		////////////////////////////////    Exo 2   /////////////////////////////////////////
		/*
		chrGPU.start();
		hipMalloc((void **) &dev_inputImg, imgBytes);
		hipMalloc((void **) &dev_output,   imgBytes);
		chrGPU.stop();
		std::cout << "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;
		// Copy data from host to device (input arrays)
		hipMemcpy(dev_inputImg, &inputImg[0], imgBytes, hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(dev_matConv), &matConv[0], matBytes, 0, hipMemcpyHostToDevice);
		// Launch kernel
		convConstCUDA<<<nb_block, nb_thread>>>(dev_inputImg, imgWidth, imgHeight, matSize, dev_output);
		// Copy data from device to host (output array)
		hipMemcpy(&output[0], dev_output, imgBytes, hipMemcpyDeviceToHost);
		*/
		////////////////////////////////////////////////////////////////////////////////////


		////////////////////////////////    Exo 3   /////////////////////////////////////////
		/*
		chrGPU.start();
		hipMalloc((void **) &dev_inputImg, imgBytes);
		hipMalloc((void **) &dev_output,   imgBytes);
		chrGPU.stop();
		std::cout << "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		// Copy data from host to device (input arrays)
		hipMemcpy(dev_inputImg, &inputImg[0], imgBytes, hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(dev_matConv), &matConv[0], matBytes, 0, hipMemcpyHostToDevice);

		hipBindTexture( 0, dev_img_1D, dev_inputImg, imgBytes);

		// Launch kernel
		conv1D_CUDA<<<nb_block, nb_thread>>>(imgWidth, imgHeight, matSize, dev_output);
		// Copy data from device to host (output array)
		hipMemcpy(&output[0], dev_output, imgBytes, hipMemcpyDeviceToHost);
		*/
		////////////////////////////////////////////////////////////////////////////////////

		////////////////////////////////    Exo 4   /////////////////////////////////////////
		size_t pitch;
		const size_t widthBytes = imgWidth * sizeof(uchar4);

		chrGPU.start();
		hipMallocPitch( &dev_inputImg, &pitch, widthBytes, imgHeight);
		hipMalloc((void **) &dev_output, imgBytes);
		chrGPU.stop();
		std::cout << "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		// Copy data from host to device (input arrays)
		hipMemcpy2D(dev_inputImg, pitch, &inputImg[0], widthBytes, widthBytes, imgHeight, hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(dev_matConv), &matConv[0], matBytes, 0, hipMemcpyHostToDevice);

		hipBindTexture2D(NULL, dev_img_2D, dev_inputImg, imgHeight, imgWidth,  pitch);

		// Launch kernel
		conv2D_CUDA<<<nb_block, nb_thread>>>(imgWidth, imgHeight, matSize, dev_output);
		// Copy data from device to host (output array)
		hipMemcpy(&output[0], dev_output, imgBytes, hipMemcpyDeviceToHost);
		////////////////////////////////////////////////////////////////////////////////////


		compareImages(resultCPU, output);
		// Free array on device
		hipFree(dev_output);
		hipFree(dev_inputImg);
	}
}
